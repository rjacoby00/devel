// Ryan Jacoby
// Compiled on GNU/Linux with nvcc v10.2.89


#include <hip/hip_runtime.h>
#include<iostream>

__global__ void output();

int main() {
    output <<<1, 32>>>();
    hipDeviceSynchronize();

    return 1;
}

__global__
void output() {
    int block = blockIdx.x;
    int thread = threadIdx.x;

    printf("Block: %d\tThread: %d\n", block, thread);
}