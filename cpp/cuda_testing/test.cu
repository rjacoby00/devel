// Ryan Jacoby
// Compiled on GNU/Linux with nvcc 10.2.89
// Test time with: nvprof --unified-memory-profiling off ./test
// Ran on RTX 2080 in 1.5752ms


#include <hip/hip_runtime.h>
#include<iostream>

__global__ void add(int, float *, float *);

int main() {
    int N = 1<<20;
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for(int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 1024;
    int blocks = (N + blockSize - 1) / blockSize;
    add<<<blocks, blockSize>>>(N, x, y);    // Blocks: how many batches of threads; block size: how many threads per block

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}

__global__ 
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}